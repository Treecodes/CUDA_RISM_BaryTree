#include "hip/hip_runtime.h"
#include <math.h>
#include <float.h>
#include <stdio.h>

#include "cuda_tcf_cp.h"


__global__ 
void  CUDA_TCF_CP_Lagrange(double kap, int batch_num_sources, int batch_idx_start,
    int cluster_q_start, int cluster_pts_start, int interp_order_lim,
    double *source_x, double *source_y, double *source_z, double *source_q,
    double *cluster_x, double *cluster_y, double *cluster_z, double *temporary_potential)
{
    int fid=threadIdx.x + blockDim.x * blockIdx.x;
    int cid_lim2 = interp_order_lim*interp_order_lim;
    int cid_lim3 = interp_order_lim*cid_lim2;
    if (fid < batch_num_sources * cid_lim3){
        int cid = fid/batch_num_sources;
        int j = fid-cid*batch_num_sources;
        int k1 = cid/cid_lim2; int tmp = cid - k1*cid_lim2;
        int k2 = tmp/interp_order_lim;
        int k3 = tmp%interp_order_lim;
        double cx = cluster_x[cluster_pts_start + k1];
        double cy = cluster_y[cluster_pts_start + k2];
        double cz = cluster_z[cluster_pts_start + k3];

        int jj = batch_idx_start + j;
        double dx = cx - source_x[jj];
        double dy = cy - source_y[jj];
        double dz = cz - source_z[jj];
        double r = sqrt(dx*dx + dy*dy + dz*dz);
        temporary_potential[j+batch_num_sources*cid] = source_q[jj] / r * 2. * exp(-kap * r);
    }
}

__host__
void K_CUDA_TCF_CP_Lagrange(
    int batch_num_sources, int batch_idx_start, 
    int cluster_q_start, int cluster_pts_start, int interp_order_lim,
    double *source_x, double *source_y, double *source_z, double *source_q,
    double *cluster_x, double *cluster_y, double *cluster_z, double *cluster_q,
    struct RunParams *run_params, int gpu_async_stream_id)
{
    double kap = run_params->kernel_params[0];
    hipError_t cudaErr;

    double *h_temporary_potential, *d_temporary_potential;
    int lim3 = interp_order_lim*interp_order_lim*interp_order_lim;
    cudaErr = hipHostMalloc(&h_temporary_potential, sizeof(double)*(batch_num_sources*lim3));
    if ( cudaErr != hipSuccess )
        printf("Host malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
    cudaErr = hipMalloc(&d_temporary_potential, sizeof(double)*(batch_num_sources*lim3));
    if ( cudaErr != hipSuccess )
        printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));

    int nthreads = 256;
    int nblocks = (batch_num_sources * lim3 - 1) / nthreads + 1;
    CUDA_TCF_CP_Lagrange<<<nblocks,nthreads>>>(kap, batch_num_sources, batch_idx_start,
                    cluster_q_start, cluster_pts_start, interp_order_lim,
                    source_x,  source_y,  source_z,  source_q,
                    cluster_x, cluster_y, cluster_z, d_temporary_potential);
    cudaErr = hipDeviceSynchronize();
    if ( cudaErr != hipSuccess )
        printf("Kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaErr));

    cudaErr = hipMemcpy(h_temporary_potential, d_temporary_potential,
                    sizeof(double)*(batch_num_sources*lim3), hipMemcpyDeviceToHost);
    if ( cudaErr != hipSuccess )
        printf("Device to Host MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
    for (int cid = 0; cid < lim3; cid++) {
        int ii = cluster_q_start + cid;
        for (int j = 0; j < batch_num_sources; j++) {
            cluster_q[ii] += h_temporary_potential[j+batch_num_sources*cid];
        }
        //printf("new %i %15.6e\n", cid, cluster_q[ii]);
    }

    hipFree(h_temporary_potential);
    hipFree(d_temporary_potential);

    return;
}
