#include "hip/hip_runtime.h"
#include <math.h>
#include <float.h>
#include <stdio.h>

#define SINGLE

#ifdef SINGLE
    #define FLOAT float
#else
    #define FLOAT double
#endif

#include "cuda_tcf_pp.h"
#include "device_vars.h"

hipError_t cudaErr;
hipStream_t stream[512];
double *d_potential;
double *d_cluster_q;
FLOAT *d_source_x;
FLOAT *d_source_y;
FLOAT *d_source_z;
FLOAT *d_source_q;
FLOAT *d_cluster_x;
FLOAT *d_cluster_y;
FLOAT *d_cluster_z;

// RQ - initialize streams
extern "C"
void initStream()
{
    for (int i = 0; i < 512; ++i) {
        cudaErr = hipStreamCreate(&stream[i]);
        if ( cudaErr != hipSuccess )
            printf("Stream creation failed with error \"%s\".\n", hipGetErrorString(cudaErr));
    }
}

extern "C"
void delStream()
{
    for (int i = 0; i < 512; ++i) {
        cudaErr = hipStreamDestroy(stream[i]);
        if ( cudaErr != hipSuccess )
            printf("Stream destruction failed with error \"%s\".\n", hipGetErrorString(cudaErr));
    }
}

// RL - initialize/free device memories
extern "C"
void CUDA_Setup(int call_type,
    int num_source, int num_cluster, int num_charge, int target_xyz_dim,
    FLOAT *source_x, FLOAT *source_y,  FLOAT *source_z, FLOAT *source_q,
    FLOAT *cluster_x, FLOAT *cluster_y, FLOAT *cluster_z,
    double *cluster_q, double *potential)
{
    if ( call_type == 1 || call_type == 3 ) {
        cudaErr = hipMalloc(&d_source_x, sizeof(FLOAT)*num_source);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMalloc(&d_source_y, sizeof(FLOAT)*num_source);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMalloc(&d_source_z, sizeof(FLOAT)*num_source);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMalloc(&d_source_q, sizeof(FLOAT)*num_source);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));

        cudaErr = hipMalloc(&d_potential, sizeof(double)*target_xyz_dim);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
    }

    if ( call_type == 1 ) {
        cudaErr = hipMalloc(&d_cluster_x, sizeof(FLOAT)*num_cluster);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMalloc(&d_cluster_y, sizeof(FLOAT)*num_cluster);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMalloc(&d_cluster_z, sizeof(FLOAT)*num_cluster);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));

        cudaErr = hipMalloc(&d_cluster_q, sizeof(double)*num_charge);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
    }


    if ( call_type == 1 || call_type == 3 ) {
        cudaErr = hipMemcpy(d_source_x, source_x, sizeof(FLOAT)*num_source, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMemcpy(d_source_y, source_y, sizeof(FLOAT)*num_source, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMemcpy(d_source_z, source_z, sizeof(FLOAT)*num_source, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMemcpy(d_source_q, source_q, sizeof(FLOAT)*num_source, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));

        cudaErr = hipMemset(d_potential, 0, sizeof(double)*target_xyz_dim);
        if ( cudaErr != hipSuccess )
            printf("Device Memset failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        //printf("CUDA copied data into device %d %d\n", num_source, target_xyz_dim);
    }

    if ( call_type == 1 ) {
        cudaErr = hipMemcpy(d_cluster_x, cluster_x, sizeof(FLOAT)*num_cluster, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMemcpy(d_cluster_y, cluster_y, sizeof(FLOAT)*num_cluster, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMemcpy(d_cluster_z, cluster_z, sizeof(FLOAT)*num_cluster, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));

        cudaErr = hipMemset(d_cluster_q, 0, sizeof(double)*num_charge);
        if ( cudaErr != hipSuccess )
            printf("Device Memset failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        //printf("CUDA copied data into device %d %d\n", num_cluster, num_charge);
    }
    return;
}

extern "C"
void CUDA_Free(int call_type,
    int num_charge, int target_xyz_dim,
    double *cluster_q, double *potential)
{
    // for direct sum we are done. copy potential back to host
    if ( call_type == 3 ) {
        cudaErr = hipMemcpy(potential, d_potential,
                             target_xyz_dim * sizeof(double), hipMemcpyDeviceToHost);
        if ( cudaErr != hipSuccess )
            printf("Device to Host MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));

        hipFree(d_source_x);
        hipFree(d_source_y);
        hipFree(d_source_z);
        hipFree(d_source_q);
        hipFree(d_potential);
    }

    // saving both potential and cluster_q for downpass in treecode
    // so no need to copy back to host
    if ( call_type == 1 ) {
        hipFree(d_cluster_x);
        hipFree(d_cluster_y);
        hipFree(d_cluster_z);
    }

    return;
}


__global__ 
static void CUDA_TCF_PP(
    FLOAT eta, FLOAT kap, FLOAT kap_eta_2,
    int cluster_num_sources, int cluster_idx_start,
    int target_x_low_ind, int target_y_low_ind, int target_z_low_ind,
    int target_x_high_ind, int target_y_high_ind, int target_z_high_ind,
    int target_yz_dim, int target_z_dim,
    FLOAT target_xmin, FLOAT target_ymin, FLOAT target_zmin,
    FLOAT target_xdd, FLOAT target_ydd, FLOAT target_zdd,
    FLOAT *source_x, FLOAT *source_y, FLOAT *source_z, FLOAT *source_q,
    double *potential)
{
    // ix/iy/iz always start from 0
    int ix = threadIdx.x + blockDim.x * blockIdx.x;
    int iy = threadIdx.y + blockDim.y * blockIdx.y;
    int iz = threadIdx.z + blockDim.z * blockIdx.z;

    if (ix < target_x_high_ind - target_x_low_ind + 1 &&
        iy < target_y_high_ind - target_y_low_ind + 1 &&
        iz < target_z_high_ind - target_z_low_ind + 1) {
        
        int ii = ((ix + target_x_low_ind) * target_yz_dim) +
                 ((iy + target_y_low_ind) * target_z_dim ) +
                  (iz + target_z_low_ind);

        FLOAT temporary_potential = 0.0;

        FLOAT tx = target_xmin + ix * target_xdd;
        FLOAT ty = target_ymin + iy * target_ydd;
        FLOAT tz = target_zmin + iz * target_zdd;

        for (int j=0; j < cluster_num_sources; j++){

            int jj = cluster_idx_start + j;
            FLOAT dx = tx - source_x[jj];
            FLOAT dy = ty - source_y[jj];
            FLOAT dz = tz - source_z[jj];
            FLOAT r  = sqrt(dx*dx + dy*dy + dz*dz);

            //if (r > DBL_MIN) {
            FLOAT kap_r = kap * r;
            FLOAT r_eta = r / eta;
            temporary_potential += source_q[jj] / r 
                                 *(exp(-kap_r) * erfc(kap_eta_2 - r_eta)
                                 - exp( kap_r) * erfc(kap_eta_2 + r_eta));
            //}

        }

        atomicAdd(potential+ii, (double)temporary_potential);

    }

    return;
}

__host__
void K_CUDA_TCF_PP(
    int num_source,
    int target_x_low_ind,  int target_x_high_ind,
    int target_y_low_ind,  int target_y_high_ind,
    int target_z_low_ind,  int target_z_high_ind,
    FLOAT target_xmin,    FLOAT target_ymin,    FLOAT target_zmin,
    FLOAT target_xdd,     FLOAT target_ydd,     FLOAT target_zdd,
    int target_x_dim_glob, int target_y_dim_glob, int target_z_dim_glob,
    int cluster_num_sources, int cluster_idx_start,
    struct RunParams *run_params, int stream_id)
{
    int target_yz_dim_glob = target_y_dim_glob * target_z_dim_glob;
    FLOAT kap = (FLOAT)run_params->kernel_params[0];
    FLOAT eta = (FLOAT)run_params->kernel_params[1];
    FLOAT kap_eta_2 = kap * eta / 2.0;

    int target_x_dim = target_x_high_ind - target_x_low_ind + 1;
    int target_y_dim = target_y_high_ind - target_y_low_ind + 1;
    int target_z_dim = target_z_high_ind - target_z_low_ind + 1;
    int threadsperblock = 8;
    dim3 nthreads(threadsperblock, threadsperblock, threadsperblock);
    dim3 nblocks((target_x_dim-1)/threadsperblock + 1,
                 (target_y_dim-1)/threadsperblock + 1,
                 (target_z_dim-1)/threadsperblock + 1);

    CUDA_TCF_PP<<<nblocks,nthreads,0,stream[stream_id]>>>(eta, kap, kap_eta_2,
                    cluster_num_sources, cluster_idx_start,
                    target_x_low_ind, target_y_low_ind, target_z_low_ind,
                    target_x_high_ind, target_y_high_ind, target_z_high_ind,
                    target_yz_dim_glob, target_z_dim_glob,
                    target_xmin, target_ymin, target_zmin,
                    target_xdd, target_ydd, target_zdd,
                    d_source_x, d_source_y, d_source_z, d_source_q, d_potential);

    return;

}
