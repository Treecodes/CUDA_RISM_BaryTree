#include "hip/hip_runtime.h"
#include <math.h>
#include <float.h>
#include <stdio.h>

#include "cuda_tcf_pp.h"


__global__ 
static void CUDA_TCF_PP(
    double eta, double kap, double kap_eta_2, int cluster_num_sources, int cluster_idx_start,
    int target_x_low_ind, int target_y_low_ind, int target_z_low_ind,
    int target_x_high_ind, int target_y_high_ind, int target_z_high_ind,
    int target_x_dim_glob, int target_y_dim_glob, int target_z_dim_glob,
    double target_xmin, double target_ymin, double target_zmin,
    double target_xdd, double target_ydd, double target_zdd,
    double *source_x, double *source_y, double *source_z, double *source_q,
    double *potential )
{
    int ix = threadIdx.x + blockDim.x * blockIdx.x;
    int iy = threadIdx.y + blockDim.y * blockIdx.y;
    int iz = threadIdx.z + blockDim.z * blockIdx.z;
    int target_yz_dim = target_y_dim_glob * target_z_dim_glob;

    if (ix >= target_x_low_ind && ix <= target_x_high_ind &&
        iy >= target_y_low_ind && iy <= target_y_high_ind &&
        iz >= target_z_low_ind && iz <= target_z_high_ind){

        double tx = target_xmin + (ix - target_x_low_ind) * target_xdd;
        double ty = target_ymin + (iy - target_y_low_ind) * target_ydd;
        double tz = target_zmin + (iz - target_z_low_ind) * target_zdd;
        int ii = (ix * target_yz_dim) + (iy * target_z_dim_glob) + iz;
        double temporary_potential = 0.0;
        for (int j=0; j < cluster_num_sources; j++){

            int jj = cluster_idx_start + j;
            double dx = tx - source_x[jj];
            double dy = ty - source_y[jj];
            double dz = tz - source_z[jj];
            double r  = sqrt(dx*dx + dy*dy + dz*dz);
            if (r > DBL_MIN) {
                double kap_r = kap *r;
                double r_eta = r / eta;
                temporary_potential += source_q[jj] / r * (exp(-kap_r) * erfc(kap_eta_2 - r_eta)
                                     - exp(kap_r) * erfc(kap_eta_2 + r_eta));
            }
        }
        potential[ii]+= temporary_potential;
    }

    return;
}


__host__
void K_CUDA_TCF_PP(
    int target_x_low_ind,  int target_x_high_ind,
    int target_y_low_ind,  int target_y_high_ind,
    int target_z_low_ind,  int target_z_high_ind,
    double target_xmin,    double target_ymin,    double target_zmin,
    double target_xdd,     double target_ydd,     double target_zdd,
    int target_x_dim_glob, int target_y_dim_glob, int target_z_dim_glob,
    int cluster_num_sources, int cluster_idx_start,
    double *source_x, double *source_y, double *source_z, double *source_q,
    struct RunParams *run_params, double *potential, int gpu_async_stream_id)
{
    double kap = run_params->kernel_params[0];
    double eta = run_params->kernel_params[1];
    double kap_eta_2 = kap * eta / 2.0;
    int threadsperblock = 4;
    dim3 nthreads(threadsperblock, threadsperblock, threadsperblock);
    dim3 nblocks((target_x_high_ind-target_x_low_ind)/threadsperblock + 1,
                 (target_y_high_ind-target_y_low_ind)/threadsperblock + 1,
                 (target_z_high_ind-target_z_low_ind)/threadsperblock + 1);

    CUDA_TCF_PP<<<nblocks,nthreads>>>(eta,kap,kap_eta_2,cluster_num_sources, cluster_idx_start,
                                    target_x_low_ind,target_y_low_ind,target_z_low_ind,
                                    target_x_high_ind,target_y_high_ind,target_z_high_ind,
                                    target_x_dim_glob,target_y_dim_glob,target_z_dim_glob,
                                    target_xmin,target_ymin,target_zmin,
                                    target_xdd,target_ydd,target_zdd,
                                    source_x, source_y, source_z, source_q,
                                    potential );

    hipError_t cudaErr;
    cudaErr = hipDeviceSynchronize();
    if ( cudaErr != hipSuccess )
        printf("Kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaErr));
    //int target_yz_dim = target_y_dim_glob * target_z_dim_glob;
    //int target_xyz_dim = target_x_dim_glob*target_yz_dim;
    //double *h_pot;
    //cudaErr = hipHostMalloc(&h_pot, target_xyz_dim*sizeof(double));
    //if ( cudaErr != hipSuccess )
    //    printf("Host malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
    //cudaErr = hipMemcpy(h_pot, potential, target_xyz_dim*sizeof(double), hipMemcpyDeviceToHost);
    //if ( cudaErr != hipSuccess )
    //    printf("Device to Host MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
    //for (int ix = target_x_low_ind; ix <= target_x_high_ind; ix++) {
    //for (int iy = target_y_low_ind; iy <= target_y_high_ind; iy++) {
    //for (int iz = target_z_low_ind; iz <= target_z_high_ind; iz++) {
    //    int ii = (ix * target_yz_dim) + (iy * target_z_dim_glob) + iz;
    //    printf("direct potential, %d %15.6e\n", ii, h_pot[ii]);
    //}
    //}
    //}
    //hipFree(h_pot);

    return;
}
