#include "hip/hip_runtime.h"
#include <math.h>
#include <float.h>
#include <stdio.h>

#include "cuda_tcf_pp.h"

__global__ 
static void CUDA_TCF_PP(
    double eta, double kap, double kap_eta_2, int cluster_num_sources, int cluster_idx_start,
    int target_x_low_ind, int target_y_low_ind, int target_z_low_ind,
    int target_x_high_ind, int target_y_high_ind, int target_z_high_ind,
    int target_yz_dim, int target_z_dim,
    double target_xmin, double target_ymin, double target_zmin,
    double target_xdd, double target_ydd, double target_zdd,
    double *source_x, double *source_y, double *source_z, double *source_q,
    double *d_potential)
{
    int ix = threadIdx.x + blockDim.x * blockIdx.x;
    int iy = threadIdx.y + blockDim.y * blockIdx.y;
    int iz = threadIdx.z + blockDim.z * blockIdx.z;

    if (ix >= target_x_low_ind && ix <= target_x_high_ind &&
        iy >= target_y_low_ind && iy <= target_y_high_ind &&
        iz >= target_z_low_ind && iz <= target_z_high_ind){

        double tx = target_xmin + (ix - target_x_low_ind) * target_xdd;
        double ty = target_ymin + (iy - target_y_low_ind) * target_ydd;
        double tz = target_zmin + (iz - target_z_low_ind) * target_zdd;
        int ii = (ix * target_yz_dim) + (iy * target_z_dim) + iz;
        double temporary_potential = 0.0;
        for (int j=0; j < cluster_num_sources; j++){

            int jj = cluster_idx_start + j;
            double dx = tx - source_x[jj];
            double dy = ty - source_y[jj];
            double dz = tz - source_z[jj];
            double r  = sqrt(dx*dx + dy*dy + dz*dz);
            if (r > DBL_MIN) {
                double kap_r = kap *r;
                double r_eta = r / eta;
                temporary_potential += source_q[jj] / r * (exp(-kap_r) * erfc(kap_eta_2 - r_eta)
                                     - exp(kap_r) * erfc(kap_eta_2 + r_eta));
            }
        }
        d_potential[ii] = temporary_potential;
        printf("kernel potential %10d %15.6e\n", ii, d_potential[ii]);
    }

    return;
}


__host__
void K_CUDA_TCF_PP(
    int target_x_low_ind,  int target_x_high_ind,
    int target_y_low_ind,  int target_y_high_ind,
    int target_z_low_ind,  int target_z_high_ind,
    double target_xmin,    double target_ymin,    double target_zmin,
    double target_xdd,     double target_ydd,     double target_zdd,
    int target_x_dim_glob, int target_y_dim_glob, int target_z_dim_glob,
    int cluster_num_sources, int cluster_idx_start,
    double *source_x, double *source_y, double *source_z, double *source_q,
    struct RunParams *run_params, double *potential, int gpu_async_stream_id)
{
    double kap = run_params->kernel_params[0];
    double eta = run_params->kernel_params[1];
    double kap_eta_2 = kap * eta / 2.0;

    hipError_t cudaErr;

    int target_x_dim = target_x_high_ind - target_x_low_ind + 1;
    int target_y_dim = target_y_high_ind - target_y_low_ind + 1;
    int target_z_dim = target_z_high_ind - target_z_low_ind + 1;
    int target_yz_dim = target_y_dim * target_z_dim;
    int target_xyz_dim = target_x_dim * target_yz_dim;

    double *h_potential;
    double *d_potential;
    cudaErr = hipHostMalloc(&h_potential, sizeof(double)*target_xyz_dim);
    if ( cudaErr != hipSuccess )
        printf("Host malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
    cudaErr = hipMalloc(&d_potential, sizeof(double)*target_xyz_dim);
    if ( cudaErr != hipSuccess )
        printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));

    int threadsperblock = 8;
    dim3 nthreads(threadsperblock, threadsperblock, threadsperblock);
    dim3 nblocks((target_x_dim-1)/threadsperblock + 1,
                 (target_y_dim-1)/threadsperblock + 1,
                 (target_z_dim-1)/threadsperblock + 1);
    CUDA_TCF_PP<<<nblocks,nthreads>>>(eta,kap,kap_eta_2, cluster_num_sources, cluster_idx_start,
                                    target_x_low_ind, target_y_low_ind, target_z_low_ind,
                                    target_x_high_ind, target_y_high_ind, target_z_high_ind,
                                    target_yz_dim, target_z_dim,
                                    target_xmin, target_ymin, target_zmin,
                                    target_xdd, target_ydd, target_zdd,
                                    source_x, source_y, source_z, source_q, d_potential);
    cudaErr = hipDeviceSynchronize();
    if ( cudaErr != hipSuccess )
        printf("Kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaErr));

    cudaErr = hipMemcpy(h_potential, d_potential,
                         target_xyz_dim * sizeof(double), hipMemcpyDeviceToHost);
    if ( cudaErr != hipSuccess )
        printf("Device to Host MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));

    int target_yz_dim_glob = target_y_dim_glob * target_z_dim_glob;
    //printf("grid block x low/high %d %d\n", target_x_low_ind, target_x_high_ind);
    //printf("grid block y low/high %d %d\n", target_y_low_ind, target_y_high_ind);
    //printf("grid block z low/high %d %d\n", target_z_low_ind, target_z_high_ind);
    for (int ix_glob = target_x_low_ind; ix_glob <= target_x_high_ind; ix_glob++) {
    for (int iy_glob = target_y_low_ind; iy_glob <= target_y_high_ind; iy_glob++) {
    for (int iz_glob = target_z_low_ind; iz_glob <= target_z_high_ind; iz_glob++) {
        int ii_glob = (ix_glob * target_yz_dim_glob) + (iy_glob * target_z_dim_glob) + iz_glob;
        int ix = ix_glob - target_x_low_ind;
        int iy = iy_glob - target_y_low_ind;
        int iz = iz_glob - target_z_low_ind; 
        int ii = (ix * target_yz_dim) + (iy * target_z_dim ) + iz;
        potential[ii_glob] += h_potential[ii];
        //printf("direct potential, %d %15.6e\n", ii_glob, h_potential[ii]);
    }
    }
    }
    hipFree(h_potential);
    hipFree(d_potential);

    return;
}
