#include "hip/hip_runtime.h"
#include <math.h>
#include <float.h>
#include <stdio.h>

//#define SINGLE

#ifdef SINGLE
    #define FLOAT float
#else
    #define FLOAT double
#endif

#include "cuda_tcf_pp.h"

// RQ: Make streams global
hipError_t cudaErr;
hipStream_t stream[4];

// RQ - initialize streams
extern "C"
void initStream() {
    for (int i = 0; i < 4; ++i) {
        cudaErr = hipStreamCreate(&stream[i]);
        if ( cudaErr != hipSuccess )
            printf("Stream creation failed with error \"%s\".\n", hipGetErrorString(cudaErr));
    }
}

extern "C"
void delStream() {
    for (int i = 0; i < 4; ++i) {
        cudaErr = hipStreamDestroy(stream[i]);
        if ( cudaErr != hipSuccess )
            printf("Stream destruction failed with error \"%s\".\n", hipGetErrorString(cudaErr));
    }
}


__global__ 
static void CUDA_TCF_PP(
    FLOAT eta, FLOAT kap, FLOAT kap_eta_2,
    int cluster_num_sources, int cluster_idx_start,
    int target_x_low_ind, int target_y_low_ind, int target_z_low_ind,
    int target_x_high_ind, int target_y_high_ind, int target_z_high_ind,
    int target_yz_dim, int target_z_dim,
    FLOAT target_xmin, FLOAT target_ymin, FLOAT target_zmin,
    FLOAT target_xdd, FLOAT target_ydd, FLOAT target_zdd,
    FLOAT *source_x, FLOAT *source_y, FLOAT *source_z, FLOAT *source_q,
    FLOAT *d_potential)
{
    int ix = threadIdx.x + blockDim.x * blockIdx.x;
    int iy = threadIdx.y + blockDim.y * blockIdx.y;
    int iz = threadIdx.z + blockDim.z * blockIdx.z;

    if (ix >= target_x_low_ind && ix <= target_x_high_ind &&
        iy >= target_y_low_ind && iy <= target_y_high_ind &&
        iz >= target_z_low_ind && iz <= target_z_high_ind){

        int ii = (ix * target_yz_dim) + (iy * target_z_dim) + iz;
        // RQ check
        if (ii < 50)
            printf("RQ inside kernel, input d_potential: %d\t\t%f\n", ii, d_potential[ii]);

        FLOAT temporary_potential = 0.0;

        FLOAT tx = target_xmin + (ix - target_x_low_ind) * target_xdd;
        FLOAT ty = target_ymin + (iy - target_y_low_ind) * target_ydd;
        FLOAT tz = target_zmin + (iz - target_z_low_ind) * target_zdd;

        for (int j=0; j < cluster_num_sources; j++){

            int jj = cluster_idx_start + j;
            FLOAT dx = tx - source_x[jj];
            FLOAT dy = ty - source_y[jj];
            FLOAT dz = tz - source_z[jj];
            FLOAT r  = sqrt(dx*dx + dy*dy + dz*dz);

            if (r > DBL_MIN) {
                FLOAT kap_r = kap * r;
                FLOAT r_eta = r / eta;
                temporary_potential += source_q[jj] / r 
                                     *(exp(-kap_r) * erfc(kap_eta_2 - r_eta)
                                     - exp( kap_r) * erfc(kap_eta_2 + r_eta));
            }

        }
        d_potential[ii] += temporary_potential;
        // RQ check
        if (ii < 50)
            printf("output potential, %d temp %15.6e d_pot %15.6e\n", ii, temporary_potential, d_potential[ii]);
    }

    return;
}


__host__
void K_CUDA_TCF_PP(
    int call_type,         int num_source,
    int target_x_low_ind,  int target_x_high_ind,
    int target_y_low_ind,  int target_y_high_ind,
    int target_z_low_ind,  int target_z_high_ind,
    FLOAT target_xmin,    FLOAT target_ymin,    FLOAT target_zmin,
    FLOAT target_xdd,     FLOAT target_ydd,     FLOAT target_zdd,
    int target_x_dim_glob, int target_y_dim_glob, int target_z_dim_glob,
    int cluster_num_sources, int cluster_idx_start,
    FLOAT *source_x, FLOAT *source_y, FLOAT *source_z, FLOAT *source_q,
    struct RunParams *run_params, double *potential, int stream_id)
{
    int target_yz_dim_glob = target_y_dim_glob * target_z_dim_glob;
    int target_xyz_dim = target_x_dim_glob * target_yz_dim_glob;

    FLOAT *d_source_x;
    FLOAT *d_source_y;
    FLOAT *d_source_z;
    FLOAT *d_source_q;
    FLOAT *d_potential;

    //printf("TCF_PP received call_type: %d\n", call_type);

    // RQ test
    if ( call_type == 1 || call_type == 3 ) {
        

        cudaErr = hipMalloc(&d_source_x, sizeof(FLOAT)*num_source);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMalloc(&d_source_y, sizeof(FLOAT)*num_source);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMalloc(&d_source_z, sizeof(FLOAT)*num_source);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMalloc(&d_source_q, sizeof(FLOAT)*num_source);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));

        cudaErr = hipMalloc(&d_potential, sizeof(FLOAT)*target_xyz_dim);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));

        // RQ Check
        printf("num_souirce: %d\n", num_source);
        //for (int i = 0; i < num_source; i++) {
        //    printf("RQ source_q %f\n", source_q[i]);
        //}


        cudaErr = hipMemcpy(d_source_x, source_x, sizeof(FLOAT)*num_source, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMemcpy(d_source_y, source_y, sizeof(FLOAT)*num_source, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMemcpy(d_source_z, source_z, sizeof(FLOAT)*num_source, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMemcpy(d_source_q, source_q, sizeof(FLOAT)*num_source, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));

        cudaErr = hipMemcpy(d_potential, potential, sizeof(FLOAT)*target_xyz_dim, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        //printf("CUDA copied data into device %d %d\n", num_source, target_xyz_dim);
    }

    FLOAT kap = (FLOAT)run_params->kernel_params[0];
    FLOAT eta = (FLOAT)run_params->kernel_params[1];
    FLOAT kap_eta_2 = kap * eta / 2.0;

    int target_x_dim = target_x_high_ind - target_x_low_ind + 1;
    int target_y_dim = target_y_high_ind - target_y_low_ind + 1;
    int target_z_dim = target_z_high_ind - target_z_low_ind + 1;
    int threadsperblock = 8;
    dim3 nthreads(threadsperblock, threadsperblock, threadsperblock);
    dim3 nblocks((target_x_dim-1)/threadsperblock + 1,
                 (target_y_dim-1)/threadsperblock + 1,
                 (target_z_dim-1)/threadsperblock + 1);

    // RQ check
    //printf("RQ check x-y-z dim: %d %d %d\n", target_x_dim, target_y_dim, target_z_dim);
    printf("RQ invoking kernel with call_type %d\n", call_type);

    // RQ - test without stream
    CUDA_TCF_PP<<<nblocks,nthreads>>>(eta, kap, kap_eta_2,
    //CUDA_TCF_PP<<<nblocks,nthreads,0,stream[stream_id]>>>(eta, kap, kap_eta_2,
                    cluster_num_sources, cluster_idx_start,
                    target_x_low_ind, target_y_low_ind, target_z_low_ind,
                    target_x_high_ind, target_y_high_ind, target_z_high_ind,
                    target_yz_dim_glob, target_z_dim_glob,
                    target_xmin, target_ymin, target_zmin,
                    target_xdd, target_ydd, target_zdd,
                    d_source_x, d_source_y, d_source_z, d_source_q, d_potential);

    // RQ
    //hipStreamSynchronize(stream[stream_id]);
    hipDeviceSynchronize();
        //printf("grid block x low/high %d %d\n", target_x_low_ind, target_x_high_ind);
        //printf("grid block y low/high %d %d\n", target_y_low_ind, target_y_high_ind);
        //printf("grid block z low/high %d %d\n", target_z_low_ind, target_z_high_ind);
        //for (int ix = target_x_low_ind; ix <= target_x_high_ind; ix++) {
        //for (int iy = target_y_low_ind; iy <= target_y_high_ind; iy++) {
        //for (int iz = target_z_low_ind; iz <= target_z_high_ind; iz++) {
        //    int ii = (ix * target_yz_dim_glob) + (iy * target_z_dim_glob ) + iz;
        //    printf("direct potential, %d %15.6e\n", ii, potential[ii]);
        //}
        //}
        //}
    // RQ test
    //if ( call_type == 2 || call_type == 3 ) {
    if ( call_type == 1 || call_type == 3 ) {

        cudaErr = hipMemcpy(potential, d_potential,
                             target_xyz_dim * sizeof(FLOAT), hipMemcpyDeviceToHost);
        if ( cudaErr != hipSuccess )
            printf("Device to Host MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));

    // RQ
    printf("RQ check potential, %d %15.6e\n", 1, potential[1]);
    printf("RQ check potential, %d %15.6e\n", 2, potential[2]);
    potential[0] = 3.1;
    //potential[1] = 3.2;
    //potential[2] = 3.3;
    potential[3] = 3.4;
    //


        // RQ test
        for (int ii = 0; ii < 50; ii++)
                printf("RQ direct potential, %d %15.6e\n", ii, potential[ii]);
        //exit(1);
        /*for (int ix = target_x_low_ind; ix <= target_x_high_ind; ix++) {
        for (int iy = target_y_low_ind; iy <= target_y_high_ind; iy++) {
        for (int iz = target_z_low_ind; iz <= target_z_high_ind; iz++) {
            int ii = (ix * target_yz_dim_glob) + (iy * target_z_dim_glob ) + iz;
            if (ii < 50)
                printf("RQ direct potential, %d %15.6e\n", ii, potential[ii]);
        }
        }
        }*/


        hipFree(d_source_x);
        hipFree(d_source_y);
        hipFree(d_source_z);
        hipFree(d_source_q);
        hipFree(d_potential);

    }

    return;

}
