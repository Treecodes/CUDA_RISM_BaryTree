#include "hip/hip_runtime.h"
#include <math.h>
#include <float.h>
#include <stdio.h>

#define SINGLE

#ifdef SINGLE
    #define FLOAT float
#else
    #define FLOAT double
#endif

#include "cuda_coulomb_pp.h"

__global__ 
static void CUDA_Coulomb_PP(
    int cluster_num_sources, int cluster_idx_start,
    int target_x_low_ind, int target_y_low_ind, int target_z_low_ind,
    int target_x_high_ind, int target_y_high_ind, int target_z_high_ind,
    int target_yz_dim, int target_z_dim,
    FLOAT target_xmin, FLOAT target_ymin, FLOAT target_zmin,
    FLOAT target_xdd, FLOAT target_ydd, FLOAT target_zdd,
    FLOAT *source_x, FLOAT *source_y, FLOAT *source_z, FLOAT *source_q,
    FLOAT *d_potential)
{
    int ix = threadIdx.x + blockDim.x * blockIdx.x;
    int iy = threadIdx.y + blockDim.y * blockIdx.y;
    int iz = threadIdx.z + blockDim.z * blockIdx.z;

    if (ix >= target_x_low_ind && ix <= target_x_high_ind &&
        iy >= target_y_low_ind && iy <= target_y_high_ind &&
        iz >= target_z_low_ind && iz <= target_z_high_ind){

        FLOAT tx = target_xmin + (ix - target_x_low_ind) * target_xdd;
        FLOAT ty = target_ymin + (iy - target_y_low_ind) * target_ydd;
        FLOAT tz = target_zmin + (iz - target_z_low_ind) * target_zdd;
        int ii = (ix * target_yz_dim) + (iy * target_z_dim) + iz;
        FLOAT temporary_potential = 0.0;
        for (int j=0;j < cluster_num_sources;j++){

            int jj = cluster_idx_start + j;
            FLOAT dx = tx - source_x[jj];
            FLOAT dy = ty - source_y[jj];
            FLOAT dz = tz - source_z[jj];
            FLOAT r  = sqrt(dx*dx + dy*dy + dz*dz);
            if (r > DBL_MIN) {
                temporary_potential += source_q[jj] / r;
            }
        }
        d_potential[ii]+= temporary_potential;
    }

    return;
}


__host__
void K_CUDA_Coulomb_PP(
    int call_type,         int num_source,
    int target_x_low_ind,  int target_x_high_ind,
    int target_y_low_ind,  int target_y_high_ind,
    int target_z_low_ind,  int target_z_high_ind,
    FLOAT target_xmin,    FLOAT target_ymin,    FLOAT target_zmin,
    FLOAT target_xdd,     FLOAT target_ydd,     FLOAT target_zdd,
    int target_x_dim_glob, int target_y_dim_glob, int target_z_dim_glob,
    int cluster_num_sources, int cluster_idx_start,
    FLOAT *source_x, FLOAT *source_y, FLOAT *source_z, FLOAT *source_q,
    struct RunParams *run_params, double *potential )
{
    FLOAT *d_source_x;
    FLOAT *d_source_y;
    FLOAT *d_source_z;
    FLOAT *d_source_q;

    printf("CUDA received call_type: %d\n", call_type);
    hipError_t cudaErr;
    if ( call_type == 1 || call_type == 3 ) {
        cudaErr = hipMalloc(&d_source_x, sizeof(FLOAT)*num_source);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMalloc(&d_source_y, sizeof(FLOAT)*num_source);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMalloc(&d_source_z, sizeof(FLOAT)*num_source);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMalloc(&d_source_q, sizeof(FLOAT)*num_source);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));

        cudaErr = hipMemcpy(d_source_x, source_x, sizeof(FLOAT)*num_source, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMemcpy(d_source_y, source_y, sizeof(FLOAT)*num_source, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMemcpy(d_source_z, source_z, sizeof(FLOAT)*num_source, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMemcpy(d_source_q, source_q, sizeof(FLOAT)*num_source, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        printf("CUDA copied data into device %d\n", num_source);
    }

    int target_x_dim = target_x_high_ind - target_x_low_ind + 1;
    int target_y_dim = target_y_high_ind - target_y_low_ind + 1;
    int target_z_dim = target_z_high_ind - target_z_low_ind + 1;
    int target_yz_dim = target_y_dim * target_z_dim;
    int target_xyz_dim = target_x_dim * target_yz_dim;

    FLOAT *h_potential;
    FLOAT *d_potential;
    cudaErr = hipHostMalloc(&h_potential, sizeof(FLOAT)*target_xyz_dim);
    if ( cudaErr != hipSuccess )
        printf("Host malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
    cudaErr = hipMalloc(&d_potential, sizeof(FLOAT)*target_xyz_dim);
    if ( cudaErr != hipSuccess )
        printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));

    int threadsperblock = 8;
    dim3 nthreads(threadsperblock, threadsperblock, threadsperblock);
    dim3 nblocks((target_x_dim-1)/threadsperblock + 1,
                 (target_y_dim-1)/threadsperblock + 1,
                 (target_z_dim-1)/threadsperblock + 1); 
    CUDA_Coulomb_PP<<<nblocks,nthreads>>>(cluster_num_sources, cluster_idx_start,
                                    0, 0, 0,
                                    target_x_dim-1, target_y_dim-1, target_z_dim-1,
                                    target_yz_dim, target_z_dim,
                                    target_xmin, target_ymin, target_zmin,
                                    target_xdd, target_ydd, target_zdd,
                                    d_source_x, d_source_y, d_source_z, d_source_q, d_potential);
    cudaErr = hipDeviceSynchronize();
    if ( cudaErr != hipSuccess )
        printf("Kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaErr));

    cudaErr = hipMemcpy(h_potential, d_potential,
                         target_xyz_dim * sizeof(FLOAT), hipMemcpyDeviceToHost);
    if ( cudaErr != hipSuccess )
        printf("Device to Host MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));

    int target_yz_dim_glob = target_y_dim_glob * target_z_dim_glob;
    //printf("grid block x low/high %d %d\n", target_x_low_ind, target_x_high_ind);
    //printf("grid block y low/high %d %d\n", target_y_low_ind, target_y_high_ind);
    //printf("grid block z low/high %d %d\n", target_z_low_ind, target_z_high_ind);
    for (int ix_glob = target_x_low_ind; ix_glob <= target_x_high_ind; ix_glob++) {
    for (int iy_glob = target_y_low_ind; iy_glob <= target_y_high_ind; iy_glob++) {
    for (int iz_glob = target_z_low_ind; iz_glob <= target_z_high_ind; iz_glob++) {
        int ii_glob = (ix_glob * target_yz_dim_glob) + (iy_glob * target_z_dim_glob) + iz_glob;
        int ix = ix_glob - target_x_low_ind;
        int iy = iy_glob - target_y_low_ind;
        int iz = iz_glob - target_z_low_ind;
        int ii = (ix * target_yz_dim) + (iy * target_z_dim ) + iz;
        potential[ii_glob] += (double)h_potential[ii];
        //printf("direct potential, %d %15.6e\n", ii_glob, h_potential[ii]);
    }
    }
    }

    hipFree(h_potential);
    hipFree(d_potential);
    if ( call_type == 1 || call_type == 3 ) {
        hipFree(d_source_x);
        hipFree(d_source_y);
        hipFree(d_source_z);
        hipFree(d_source_q);
    }

    return;
}

