#include "hip/hip_runtime.h"
#include <math.h>
#include <float.h>
#include <stdio.h>

#include "cuda_dcf_cp.h"


__global__ 
void CUDA_DCF_CP_Lagrange(double eta, int batch_num_sources, int batch_idx_start,
    int cluster_q_start, int cluster_pts_start, int interp_order_lim,
    double *source_x, double *source_y, double *source_z, double *source_q,
    double *cluster_x, double *cluster_y, double *cluster_z, double *temporary_potential)
{
    int fid=threadIdx.x + blockDim.x * blockIdx.x;
    int cid_lim2 = interp_order_lim*interp_order_lim;
    int cid_lim3 = interp_order_lim*cid_lim2;
    if (fid < batch_num_sources * cid_lim3){
        int cid = fid/batch_num_sources;
        int j = fid-cid*batch_num_sources;
        int k1 = cid/cid_lim2; int tmp = cid - k1*cid_lim2;
        int k2 = tmp/interp_order_lim;
        int k3 = tmp%interp_order_lim;
        double cx = cluster_x[cluster_pts_start + k1];
        double cy = cluster_y[cluster_pts_start + k2];
        double cz = cluster_z[cluster_pts_start + k3];

        int jj = batch_idx_start + j;
        double dx = cx - source_x[jj];
        double dy = cy - source_y[jj];
        double dz = cz - source_z[jj];
        double r = sqrt(dx*dx + dy*dy + dz*dz);
        temporary_potential[j+batch_num_sources*cid] = source_q[jj] * erf(r / eta) / r;
    }
}

__host__
void K_CUDA_DCF_CP_Lagrange(
    int call_type, int num_source, int num_cluster,
    int batch_num_sources, int batch_idx_start, 
    int cluster_q_start, int cluster_pts_start, int interp_order_lim,
    double *source_x, double *source_y, double *source_z, double *source_q,
    double *cluster_x, double *cluster_y, double *cluster_z, double *cluster_q,
    struct RunParams *run_params)
{
    double eta = run_params->kernel_params[0];
    double *d_source_x;
    double *d_source_y; 
    double *d_source_z;
    double *d_source_q;
    double *d_cluster_x;
    double *d_cluster_y;
    double *d_cluster_z;

   // printf("CUDA received call_type: %d\n", call_type);
    hipError_t cudaErr;
    if ( call_type == 1 ) {
        cudaErr = hipMalloc(&d_source_x, sizeof(double)*num_source);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMalloc(&d_source_y, sizeof(double)*num_source);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMalloc(&d_source_z, sizeof(double)*num_source);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMalloc(&d_source_q, sizeof(double)*num_source);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));

        cudaErr = hipMalloc(&d_cluster_x, sizeof(double)*num_cluster);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMalloc(&d_cluster_y, sizeof(double)*num_cluster);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMalloc(&d_cluster_z, sizeof(double)*num_cluster);
        if ( cudaErr != hipSuccess )
            printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));

        cudaErr = hipMemcpy(d_source_x, source_x, sizeof(double)*num_source, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMemcpy(d_source_y, source_y, sizeof(double)*num_source, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMemcpy(d_source_z, source_z, sizeof(double)*num_source, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMemcpy(d_source_q, source_q, sizeof(double)*num_source, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMemcpy(d_cluster_x, cluster_x, sizeof(double)*num_cluster, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMemcpy(d_cluster_y, cluster_y, sizeof(double)*num_cluster, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
        cudaErr = hipMemcpy(d_cluster_z, cluster_z, sizeof(double)*num_cluster, hipMemcpyHostToDevice);
        if ( cudaErr != hipSuccess )
            printf("Host to Device MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
     //   printf("CUDA copied data into device %d %d\n", num_source, num_cluster);
    }

    double *h_temporary_potential, *d_temporary_potential;
    int lim3 = interp_order_lim*interp_order_lim*interp_order_lim;
    cudaErr = hipHostMalloc(&h_temporary_potential, sizeof(double)*(batch_num_sources*lim3));
    if ( cudaErr != hipSuccess )
        printf("Host malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));
    cudaErr = hipMalloc(&d_temporary_potential, sizeof(double)*(batch_num_sources*lim3));
    if ( cudaErr != hipSuccess )
        printf("Device malloc failed with error \"%s\".\n", hipGetErrorString(cudaErr));

    int nthreads = 256;
    int nblocks = (batch_num_sources*lim3 - 1) / nthreads + 1;
    CUDA_DCF_CP_Lagrange<<<nblocks,nthreads>>>(eta, batch_num_sources, batch_idx_start,
                    cluster_q_start, cluster_pts_start, interp_order_lim,
                    d_source_x,  d_source_y,  d_source_z,  d_source_q,
                    d_cluster_x, d_cluster_y, d_cluster_z, d_temporary_potential);
    cudaErr = hipDeviceSynchronize();
    if ( cudaErr != hipSuccess )
        printf("Kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaErr));

    cudaErr = hipMemcpy(h_temporary_potential, d_temporary_potential,
                    sizeof(double)*(batch_num_sources*lim3), hipMemcpyDeviceToHost);
    if ( cudaErr != hipSuccess )
        printf("Device to Host MemCpy failed with error \"%s\".\n", hipGetErrorString(cudaErr));
    for (int cid = 0; cid < lim3; cid++) {
        int ii = cluster_q_start + cid;
        for (int j = 0; j < batch_num_sources; j++) {
            cluster_q[ii] += h_temporary_potential[j+batch_num_sources*cid];
        }
    }

    hipFree(h_temporary_potential);
    hipFree(d_temporary_potential);
    if ( call_type == 2 ) {
        hipFree(d_source_x);
        hipFree(d_source_y);
        hipFree(d_source_z);
        hipFree(d_source_q);
        hipFree(d_cluster_x);
        hipFree(d_cluster_y);
        hipFree(d_cluster_z);
    }

    return;
}
