#include "hip/hip_runtime.h"
#include <math.h>
#include <float.h>
#include <stdio.h>

//#define SINGLE

#ifdef SINGLE
    #define FLOAT float
#else
    #define FLOAT double
#endif

#include "cuda_dcf_cp.h"
#include "../tcf/device_vars.h"

__global__ 
void CUDA_DCF_CP_Lagrange(FLOAT eta, int batch_num_sources, int batch_idx_start,
    int cluster_q_start, int cluster_pts_start, int interp_order_lim,
    FLOAT *source_x, FLOAT *source_y, FLOAT *source_z, FLOAT *source_q,
    FLOAT *cluster_x, FLOAT *cluster_y, FLOAT *cluster_z, FLOAT *potential)
{

    int k1 = threadIdx.x + blockDim.x * blockIdx.x;
    int k2 = threadIdx.y + blockDim.y * blockIdx.y;
    int k3 = threadIdx.z + blockDim.z * blockIdx.z;

    if ( k1 >=0 && k1 < interp_order_lim &&
         k2 >=0 && k2 < interp_order_lim &&
         k3 >=0 && k3 < interp_order_lim ){

        FLOAT temporary_potential = 0.0;

        FLOAT cx = cluster_x[cluster_pts_start + k1];
        FLOAT cy = cluster_y[cluster_pts_start + k2];
        FLOAT cz = cluster_z[cluster_pts_start + k3];

        int ii = cluster_q_start + k1 * interp_order_lim*interp_order_lim + k2 * interp_order_lim + k3;
        for (int j = 0; j < batch_num_sources; j++) {

            int jj = batch_idx_start + j;
            FLOAT dx = cx - source_x[jj];
            FLOAT dy = cy - source_y[jj];
            FLOAT dz = cz - source_z[jj];
            FLOAT r = sqrt(dx*dx + dy*dy + dz*dz);

         if (r > DBL_MIN) {
            temporary_potential += source_q[jj] * erf(r / eta) / r;
         }
        }
        potential[ii] += temporary_potential;
    }
 return;
}

__host__
void K_CUDA_DCF_CP_Lagrange(
    int call_type, int num_source, int num_cluster,
    int batch_num_sources, int batch_idx_start, 
    int cluster_q_start, int cluster_pts_start, int interp_order_lim,
    FLOAT *source_x, FLOAT *source_y, FLOAT *source_z, FLOAT *source_q,
    FLOAT *cluster_x, FLOAT *cluster_y, FLOAT *cluster_z, double *cluster_q,
    struct RunParams *run_params, int stream_id)
{
    FLOAT eta = (FLOAT)run_params->kernel_params[0];

    //printf("CUDA received call_type: %d\n", call_type);

    int threadsperblock = 8;
    dim3 nthreads(threadsperblock, threadsperblock, threadsperblock);
    dim3 nblocks((interp_order_lim-1)/threadsperblock + 1,
                 (interp_order_lim-1)/threadsperblock + 1,
                 (interp_order_lim-1)/threadsperblock + 1);

    CUDA_DCF_CP_Lagrange<<<nblocks,nthreads,0,stream[stream_id]>>>(eta, batch_num_sources, batch_idx_start,
                    cluster_q_start, cluster_pts_start, interp_order_lim,
                    d_source_x,  d_source_y,  d_source_z,  d_source_q,
                    d_cluster_x, d_cluster_y, d_cluster_z, d_cluster_q);


    return;
}
